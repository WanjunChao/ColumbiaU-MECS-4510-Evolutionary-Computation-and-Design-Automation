
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vecAdd(void)
{
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

void vecAdd_wrapper() {

    // Execute the kernel
    vecAdd <<< 10, 10 >>> ();
    hipDeviceSynchronize();
    printf("Hello World!\n");
}
